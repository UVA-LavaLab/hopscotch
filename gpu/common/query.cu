/*******************************************************************************
 *
 * File: query.cu
 * Description: Query device properties of all the cuda gpus present in the
 *              system and write them in JSON format.
 * 
 * Author: Alif Ahmed
 * Date: Aug 06, 2019
 *
 ******************************************************************************/
#include "common.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

/**
 * Query some of the device properties. Writes to stdout in JSON format.
 */
void printDeviceInfo(int device) {
	hipDeviceProp_t prop;
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, device));
	cout << "    {" << endl;
	cout << "      \"name\": \"" 				<< prop.name << "\",\n";
	cout << "      \"totalGlobalMem\": " 	<< prop.totalGlobalMem << ",\n";
	cout << "      \"clockRate\": " 		<< prop.clockRate << ",\n";
	cout << "      \"computeCapability\": \"" << prop.major << "." << prop.minor << "\",\n";
	cout << "      \"multiProcessorCount\": "	<< prop.multiProcessorCount	<< ",\n";
	cout << "      \"memoryClockRate\": "	<< prop.memoryClockRate	<< ",\n";
	cout << "      \"memoryBusWidth\": "	<< prop.memoryBusWidth	<< ",\n";
	cout << "      \"warpSize\": "	<< prop.warpSize	<< "\n";
	cout << "    }";
}

int main() {
	int deviceCount;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount));
	cout << "{\n" << "  \"cudaDevices\": [\n";
	for(int i = 0; i < deviceCount; ++i) {
		if(i) cout << ",\n";
		printDeviceInfo(i);
	}
	cout << "\n  ]\n}\n";
	return 0;
}
