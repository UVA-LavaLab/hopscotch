#include "common.h"
#include <string>
#include <iostream>

using namespace std;

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
